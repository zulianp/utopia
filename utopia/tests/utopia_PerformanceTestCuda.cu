#include "utopia_PerformanceTestCuda.hpp"
#include "utopia_CUDABackendTPL.hpp"
#include "utopia_CUDAMatrix.hpp"
#include "utopia_CUDAVector.hpp"

namespace utopia{

//   int KK=7;
  
   static int SIZES[7] = {16, 32, 64, 128, 256, 512, 1024};
 
   void test_CUDA(const std::string &experiment_name)
  {
          std::cout << "Running " << experiment_name << std::endl;
          int times = 10;
	  CUDAVector<double> v;
	  CUDAMatrix<double> m;
	  CUDAVector<double> resv;
	  CUDAMatrix<double> resm; 

	  for (int sz=0; sz < 7; sz ++) {

            double * v_h = (double *) malloc (SIZES[sz]*sizeof(double));

	    m.values.reserve(SIZES[sz]*SIZES[sz]);
	    m.values.resize(SIZES[sz]*SIZES[sz]);

	    v.values.reserve(SIZES[sz]); 
	    v.values.resize(SIZES[sz]);
 
	    resv.values.reserve(SIZES[sz]); 
	    resv.values.resize(SIZES[sz]);
 
	    resm.values.reserve(SIZES[sz]*SIZES[sz]);
	    resm.values.resize(SIZES[sz]*SIZES[sz]);

            //fill matrices and vectors
            cuda_double::build_values(SIZES[sz], 2, v);
            cuda_double::build_values(SIZES[sz], SIZES[sz], 2, m);
            
            double * v_ptr = thrust::raw_pointer_cast(&v.values[0]);
            

            //thrust::copy(v_ptr, v_ptr+SIZES[sz], v_h);            
            hipMemcpy(v_h, v_ptr, SIZES[sz]*sizeof(double), hipMemcpyDeviceToHost);
            
  //          for (int j=0; j<SIZES[sz]; ++j) std::cout << "v_h[0] = " << v_h[j] << std::endl; 

	    utopia::Chrono c;
	    //test matrix-vector multiplication
	    c.start();
	    for (int i = 0; i < times; ++i) {
	      cuda_double::mat_vec_mul( m, v, resv);
	    }
	    c.stop();
            std::cout << "WARNING we are performing the same opeartion for "<< times <<" times\n";

	    std::cout << "CUDA Matrix-Vector Results\n";
            std::cout << "Size Problem\n";
            std::cout << "Size Matrix Array==>"<<SIZES[sz]*SIZES[sz]<<"\n";
            std::cout << "Size Problem\n";
            std::cout << "Size Vector Array==>"<<SIZES[sz]<<"\n";
	    c.describe(std::cout);

	    // test matrix-matrix multiplication
	    c.start();
	    for (int i = 0; i < times; ++i) {
	      cuda_double::mat_mat_mul( m, m, SIZES[sz], resm);
	    }
	    c.stop();
	    std::cout << "CUDA Matrix-Matrix Results\n";
            std::cout << "Size Problem\n";
            std::cout << "Size Matrix Array==>"<<SIZES[sz]*SIZES[sz]<<"\n";
	    c.describe(std::cout);
	  

	  }

  }

   void test_CPU(const std::string &experiment_name)
  {
   int times = 10;
    std::cout << "Running " << experiment_name << std::endl;  
      for (int sz=0; sz < 7; sz ++) {
        int length_v=SIZES[sz];
        int length_m=SIZES[sz]*SIZES[sz]; 
        double v[length_v];
        double m[length_m];
        double res_v[length_v];
        double res_m[length_m];
        for (int k=0; k<length_v; ++k) v[k]=2;
        for (int k=0; k<length_m; ++k) m[k]=2;
        utopia::Chrono c;
        c.start();
        for (int ii = 0; ii < times; ++ii) {
        //     std::cout <<"ciao"<<std::endl;
             for (int l=0; l<length_v; ++l){
                 double res=0;
                  for (int k=0; k<length_v; ++k){
                        double m_x=m[l*length_v+k];
                        double v_x=v[k];
                        res+= m_x*v_x;
                      }
                      res_v[l]=res;
                 }
        }
       c.stop();
       std::cout << "WARING we are performing the same opeartion for "<< times <<" times\n";
       
       std::cout << "CPU Matrix-Vector Results\n";
       std::cout << "Size Problem\n";
       std::cout << "Size Matrix Array==>"<<SIZES[sz]*SIZES[sz]<<"\n";
       std::cout << "Size Problem\n";
       std::cout << "Size Vector Array==>"<<SIZES[sz]<<"\n";
       c.describe(std::cout);
      
//       for (int j=0; j<SIZES[sz]; ++j) std::cout << "v_res[0] = " << res_v[j] << std::endl;       

       c.start();
      for (int ii = 0; ii < times; ++ii){
           for (int l=0; l<length_v; ++l){
                 for (int ll=0; ll<length_v; ++ll){   
                      double res=0;
                      for(int k=0; k<length_v; ++k){
                           double m_x=m[l*length_v+k];
                           double m_y=m[k*length_v+ll];
                           res+= m_x*m_y;
                       }  
                   res_m[l*length_v+ll]=res;
               }
          }
      }
     c.stop();
     std::cout << "CPU Matrix-Matrix Results\n";
     std::cout << "Size Problem\n";
     std::cout << "Size Matrix Array==>"<<SIZES[sz]*SIZES[sz]<<"\n";
     c.describe(std::cout);
              
   }

}
  //int size = 
  //hipMalloc(&m , WIDTH*WIDTH*sizeof (int) ) ;
  //hipMalloc(&m , WIDTH*WIDTH*sizeof (int) ) 
  void run_performance_CUDA_test(){          
 
	   test_CUDA("CUDA");

           test_CPU("CPU");
     } 

}
