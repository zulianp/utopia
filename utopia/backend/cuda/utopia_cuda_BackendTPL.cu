#include "hip/hip_runtime.h"
#include "utopia_cuda_BackendTPL.hpp"

#include <thrust/inner_product.h>
#include <utility>

#include "utopia_cuda_Error.hpp"

namespace utopia {

	namespace cuda_generic {
		template<typename T>
		__global__ void build_identity(const int rows, const int cols, T *values)
		{
			int id_x = blockIdx.x * blockDim.x + threadIdx.x;
			int id_y = blockIdx.y * blockDim.y + threadIdx.y;

			if(id_x < rows && id_y < cols) {
				if(id_x == id_y) {
					values[id_x * cols + id_y] = 1;
				} else {
					values[id_x * cols + id_y] = 0;
				}
			}
		}

		template<typename T>
		__global__ void mat_vec_mul(const int rows, const int cols, const T *mat_left, const T *vec_right, T *result)
		{
			int id_x = blockIdx.x * blockDim.x + threadIdx.x;
			if(id_x < rows) {
				
				const int offset_i = id_x * cols;

				T prod = 0;
				for(int j = 0; j < cols; ++j) {
					prod += mat_left[offset_i + j] * vec_right[j];
				}

				// __synchthreads();
				result[id_x] = prod;
			}
		}

		std::pair<dim3, dim3> get_sizes_2(const int n_x, const int n_y)
		{
			// dim3 n_blocks( ceil(n_x/22.), ceil(n_y/22.) );
			// dim3 n_threads_x_block(n_x/n_blocks.x + 1, 
			// 					   n_y/n_blocks.y + 1);

			dim3 n_blocks(n_x, n_y);
			dim3 n_threads_x_block(1, 1);

			return std::make_pair(n_blocks, n_threads_x_block);
		}

		std::pair<dim3, dim3> get_sizes_1(const int n_x)
		{
			dim3 n_blocks(ceil(n_x/512.));
			dim3 n_threads_x_block(ceil(double(n_x)/n_blocks.x));
			return std::make_pair(n_blocks, n_threads_x_block);
		}
	}


	namespace cuda_double {

		template<class Tensor>
		static void describe_values(const Tensor &t) 
		{
			thrust::copy(t.values.begin(), t.values.end(), std::ostream_iterator<double>(std::cout, "\n"));
		}

		void describe(const CUDAMatrix &m)
		{
			describe_values(m);
		}

		void describe(const CUDAVector &v)
		{
			describe_values(v);
		}

		void build_identity(const int rows, const int cols, CUDAMatrix &m)
		{
			m.values.resize(rows * cols);
			m.rows = rows;
			m.cols = cols;

			double *raw_ptr = thrust::raw_pointer_cast(&m.values[0]);
			dim3 n_blocks, n_threads_x_block;
			std::pair<dim3, dim3> s = cuda_generic::get_sizes_2(rows, cols);

			cuda_generic::build_identity<double><<<s.first, s.second>>>(rows, cols, raw_ptr);
		}
		
		void build_values(const int n, const double value, CUDAVector &v)
		{
			v.values.resize(n);
			thrust::fill(v.values.begin(), v.values.end(), value);
		}

		double dot(const CUDAVector &left, const CUDAVector &right)
		{
			return thrust::inner_product(left.values.begin(), left.values.end(), right.values.begin(), 0.0);
		}

		void mat_vec_mul(const CUDAMatrix &left, const CUDAVector &right, CUDAVector &result)
		{
			result.values.resize(left.rows);

			dim3 n_blocks, n_threads_x_block;
			
			std::pair<dim3, dim3> s = cuda_generic::get_sizes_1(left.rows);

			cuda_generic::mat_vec_mul<double><<<s.first, s.second>>>(
				left.rows,
				left.cols,
				thrust::raw_pointer_cast(&left.values[0]),
				thrust::raw_pointer_cast(&right.values[0]),
				thrust::raw_pointer_cast(&result.values[0])
				);

			CUDAError::CheckLastError();
		}
	}
}

