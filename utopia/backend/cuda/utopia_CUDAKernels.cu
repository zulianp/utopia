
#include <hip/hip_runtime.h>
//#ifdef WITH_CUDA
//
//#ifndef UTOPIA_CUDA_KERNELS_H
//#define UTOPIA_CUDA_KERNELS_H
//
//namespace utopia {
//    namespace cuda {
//        template<typename T, class SizeType, class Operation>
//        __global__ void apply(const T *left, const T *right, T *result, SizeType n) {
//            Operation op;
//            const unsigned int nThreads = gridDim.x * blockDim.x;
//            for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += nThreads) {
//                result[i] = op(left[i], right[i]);
//            }
//        }
//
//        template<typename T>
//        class Add {
//        public:
//            __host__ __device__ T operator()(const T &left, const T &right) const {
//                return left + right;
//            }
//        };
//
//        template<typename T, class SizeType>
//        __host__ void sum(const T *left, const T *right, T *result, SizeType n) {
//            apply<T, SizeType, Add<T> > <<<n, 1>>>(left, right, n);
//        }
//    }
//}
//
//#endif //UTOPIA_CUDA_KERNELS_H
//#endif //WITH_CUDA
//
//
//
//
